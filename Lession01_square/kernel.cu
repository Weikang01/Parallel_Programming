﻿
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void square(int *d_out, int *d_in)
{
    int idx = threadIdx.x;
    int i = d_in[idx];
    d_out[idx] = i * i;
}

int main()
{
    const int ARRAY_SIZE = 64;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);

    // generate the input array on the host
    int h_in[ARRAY_SIZE];
    for (int i = 0; i < ARRAY_SIZE; i++)
    {
        h_in[i] = i;
    }
    int h_out[ARRAY_SIZE];

    // declare GPU memory pointers
    int* d_in;
    int* d_out;

    // allocate GPU memory
    hipMalloc((void**)&d_in, ARRAY_BYTES);
    hipMalloc((void**)&d_out, ARRAY_BYTES);

    // transfer the array to the GPU
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyKind::hipMemcpyHostToDevice);

    // launch the kernel
    square <<<1, ARRAY_SIZE >>> (d_out, d_in);

    // copy back the result array to the CPU
    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyKind::hipMemcpyDeviceToHost);

    // print out the resulting array
    for (int i = 0; i < ARRAY_SIZE; i++)
    {
        printf("%d", h_out[i]);
        printf(((i % 4) != 3) ? "\t" : "\n");
    }

    // free GPU memory allocation
    hipFree(d_in);
    hipFree(d_out);

    return 0;
}
